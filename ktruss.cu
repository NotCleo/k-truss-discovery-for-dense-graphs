#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

// CSR format for the graph
struct Graph {
    int* rowPtr;  // Row pointers
    int* colIdx;  // Column indices
    int numNodes;
    int numEdges;
    std::vector<std::pair<int, int>> edges; // Explicit edge list
};

// Kernel to count triangles per edge
__global__ void countTriangles(int* rowPtr, int* colIdx, int* edgeSrc, int* edgeDst, int* support, int numEdges) {
    int edgeIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (edgeIdx >= numEdges) return;

    int u = edgeSrc[edgeIdx];
    int v = edgeDst[edgeIdx];
    int uStart = rowPtr[u], uEnd = rowPtr[u + 1];
    int vStart = rowPtr[v], vEnd = rowPtr[v + 1];
    int count = 0;

    // Use two-pointer technique for efficiency
    int i = uStart, j = vStart;
    while (i < uEnd && j < vEnd) {
        int uNeighbor = colIdx[i];
        int vNeighbor = colIdx[j];
        if (uNeighbor == v) { i++; continue; } // Skip v in u's list
        if (vNeighbor == u) { j++; continue; } // Skip u in v's list
        if (uNeighbor == vNeighbor) {
            count++;
            i++;
            j++;
        } else if (uNeighbor < vNeighbor) {
            i++;
        } else {
            j++;
        }
    }
    support[edgeIdx] = count;
}

// Main k-truss function
void kTruss(Graph& g, int k) {
    // Populate edge arrays
    std::vector<int> edgeSrc(g.numEdges), edgeDst(g.numEdges);
    for (int i = 0; i < g.numEdges; i++) {
        edgeSrc[i] = g.edges[i].first;
        edgeDst[i] = g.edges[i].second;
    }

    // Debug: Print edges
    printf("Debug - Edge List:\n");
    for (int i = 0; i < g.numEdges; i++) {
        printf("Edge %d: %d-%d\n", i, edgeSrc[i], edgeDst[i]);
    }

    // Allocate GPU memory
    int *d_rowPtr, *d_colIdx, *d_edgeSrc, *d_edgeDst, *d_support;
    hipMalloc(&d_rowPtr, (g.numNodes + 1) * sizeof(int));
    hipMalloc(&d_colIdx, (g.rowPtr[g.numNodes]) * sizeof(int));
    hipMalloc(&d_edgeSrc, g.numEdges * sizeof(int));
    hipMalloc(&d_edgeDst, g.numEdges * sizeof(int));
    hipMalloc(&d_support, g.numEdges * sizeof(int));

    hipMemcpy(d_rowPtr, g.rowPtr, (g.numNodes + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_colIdx, g.colIdx, (g.rowPtr[g.numNodes]) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edgeSrc, edgeSrc.data(), g.numEdges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edgeDst, edgeDst.data(), g.numEdges * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threads = 256;
    int blocks = (g.numEdges + threads - 1) / threads;
    countTriangles<<<blocks, threads>>>(d_rowPtr, d_colIdx, d_edgeSrc, d_edgeDst, d_support, g.numEdges);
    hipDeviceSynchronize();

    // Check for CUDA errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) printf("CUDA Error after kernel launch: %s\n", hipGetErrorString(err));
    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("CUDA Error after synchronize: %s\n", hipGetErrorString(err));

    // Copy supports back to host
    std::vector<int> support(g.numEdges);
    hipMemcpy(support.data(), d_support, g.numEdges * sizeof(int), hipMemcpyDeviceToHost);
    err = hipGetLastError();
    if (err != hipSuccess) printf("CUDA Error after memcpy: %s\n", hipGetErrorString(err));

    // Debug: Print supports before pruning
    printf("Debug - Supports Before Pruning:\n");
    for (int i = 0; i < g.numEdges; i++) {
        printf("Edge %d: %d-%d, Support: %d\n", i, edgeSrc[i], edgeDst[i], support[i]);
    }

    // Iterative pruning for k-truss
    for (int iter = 0; iter < g.numEdges; iter++) {
        bool changed = false;
        for (int i = 0; i < g.numEdges; i++) {
            if (support[i] >= 0 && support[i] < k - 2) {
                support[i] = -1; // Mark for removal
                changed = true;
            }
        }
        if (!changed) break;
    }

    // Print remaining edges
    printf("3-Truss Edges:\n");
    for (int i = 0; i < g.numEdges; i++) {
        if (support[i] >= k - 2) {
            printf("Edge %d: %d-%d, Support: %d\n", i, edgeSrc[i], edgeDst[i], support[i]);
        }
    }

    hipFree(d_rowPtr); hipFree(d_colIdx); hipFree(d_edgeSrc); hipFree(d_edgeDst); hipFree(d_support);
}

int main() {
    // Hardcoded small graph: 4 nodes, 5 edges
    int numNodes = 4;
    int numEdges = 5;
    std::vector<int> rowPtr = {0, 2, 4, 6, 7}; // CSR row pointers
    std::vector<int> colIdx = {1, 2, 0, 2, 0, 1, 3, 3}; // CSR column indices (undirected)
    std::vector<std::pair<int, int>> edges = {{0, 1}, {0, 2}, {1, 2}, {1, 3}, {2, 3}}; // Explicit edge list

    Graph g = {rowPtr.data(), colIdx.data(), numNodes, numEdges, edges};
    kTruss(g, 3); // Find 3-truss
    return 0;
}
